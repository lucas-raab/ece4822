#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include "p01.h"
#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void mmult_kernel(float* mat3, float* mat1, float* mat2, long nrows, long ncols, long start_row, long end_row) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + start_row;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < end_row && col < nrows) {
        float sum = 0.0f;
        for (int k = 0; k < ncols; k++) {
            sum += mat1[(row - start_row) * ncols + k] * mat2[k * nrows + col];
        }
        mat3[(row - start_row) * nrows + col] = sum;
    }
}

int main(int argc, char** argv) {
    if (argc != 4) {
        fprintf(stderr, "ERROR, p01 requires 3 input arguments\n");
        return -1;
    }

    char *out;
    long nrows = strtol(argv[1], &out, 10);
    if (argv[1] == out) {
        fprintf(stderr, "ERROR, nrows not a number\n");
        return -1;
    }

    long ncols = strtol(argv[2], &out, 10);
    if (argv[2] == out) {
        fprintf(stderr, "ERROR, ncols not a number\n");
        return -1;
    }

    long niter = strtol(argv[3], &out, 10);
    if (argv[3] == out) {
        fprintf(stderr, "ERROR, niter not a number\n");
        return -1;
    }

    float *mat1, *mat2, *mat3;

    mat1 = (float*)malloc(sizeof(float) * nrows * ncols);
    mat2 = (float*)malloc(sizeof(float) * nrows * ncols);
    mat3 = (float*)malloc(sizeof(float) * nrows * nrows);

    // Initialize host arrays
    genmat(mat1, nrows, ncols);
    genmat(mat2, ncols, nrows);

    // Get the number of available GPUs
    int num_gpus;
    hipError_t err = hipGetDeviceCount(&num_gpus);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    printf("Number of GPUs: %d\n", num_gpus);

    // Allocate memory for device pointers and CUDA streams
    float **d_mat1 = (float**)malloc(num_gpus * sizeof(float*));
    float **d_mat2 = (float**)malloc(num_gpus * sizeof(float*));
    float **d_mat3 = (float**)malloc(num_gpus * sizeof(float*));
    hipStream_t *streams = (hipStream_t*)malloc(num_gpus * sizeof(hipStream_t));

    if (d_mat1 == NULL || d_mat2 == NULL || d_mat3 == NULL || streams == NULL) {
        fprintf(stderr, "Failed to allocate host memory for device pointers or streams\n");
        return -1;
    }

    // Calculate the number of rows per GPU
    long rows_per_gpu = nrows / num_gpus;
    long remainder = nrows % num_gpus;

    for (int i = 0; i < num_gpus; i++) {
        printf("gpu : %d\n", i);
        err = hipSetDevice(i);
        if (err != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed for GPU %d: %s\n", i, hipGetErrorString(err));
            continue;
        }

        long start_row = i * rows_per_gpu + (i < remainder ? i : remainder);
        long end_row = start_row + rows_per_gpu + (i < remainder ? 1 : 0);
        long gpu_nrows = end_row - start_row;

        err = hipMalloc((void**)&d_mat1[i], sizeof(float) * gpu_nrows * ncols);
        if (err != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for d_mat1[%d]: %s\n", i, hipGetErrorString(err));
            continue;
        }

        err = hipMalloc((void**)&d_mat2[i], sizeof(float) * nrows * ncols);
        if (err != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for d_mat2[%d]: %s\n", i, hipGetErrorString(err));
            continue;
        }

        err = hipMalloc((void**)&d_mat3[i], sizeof(float) * gpu_nrows * nrows);
        if (err != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for d_mat3[%d]: %s\n", i, hipGetErrorString(err));
            continue;
        }

        err = hipStreamCreate(&streams[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "hipStreamCreate failed for GPU %d: %s\n", i, hipGetErrorString(err));
            continue;
        }

        err = hipMemcpyAsync(d_mat1[i], mat1 + start_row * ncols, sizeof(float) * gpu_nrows * ncols, hipMemcpyHostToDevice, streams[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync failed for d_mat1[%d]: %s\n", i, hipGetErrorString(err));
            continue;
        }

        err = hipMemcpyAsync(d_mat2[i], mat2, sizeof(float) * nrows * ncols, hipMemcpyHostToDevice, streams[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync failed for d_mat2[%d]: %s\n", i, hipGetErrorString(err));
            continue;
        }

        // Get the maximum number of threads per block for this GPU
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess) {
            fprintf(stderr, "hipGetDeviceProperties failed for GPU %d: %s\n", i, hipGetErrorString(err));
            continue;
        }
        int max_threads_per_block = prop.maxThreadsPerBlock;

        // Calculate optimal block and grid sizes
        int block_size = 16;
        dim3 blockSize(block_size, block_size);
        dim3 gridSize((nrows + blockSize.x - 1) / blockSize.x,
                      (gpu_nrows + blockSize.y - 1) / blockSize.y);

        printf("GPU %d: Start Row: %ld, End Row: %ld, Block Size: %d\n", i, start_row, end_row, block_size);
    }


    clock_t start, end;
    double elapsed_time = 0;

    int iter = niter;
    start = clock();

    printf("starting\n");
    for (int i = 0; i < num_gpus; i++) {
            hipSetDevice(i);
            
            long start_row = i * rows_per_gpu + (i < remainder ? i : remainder);
            long end_row = start_row + rows_per_gpu + (i < remainder ? 1 : 0);
            long gpu_nrows = end_row - start_row;

            // Get the maximum number of threads per block for this GPU
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
            int max_threads_per_block = prop.maxThreadsPerBlock;

            // Calculate optimal block and grid sizes
            int block_size = (int)sqrt(max_threads_per_block);
            dim3 blockSize(block_size, block_size);
            dim3 gridSize((nrows + blockSize.x - 1) / blockSize.x,
                          (gpu_nrows + blockSize.y - 1) / blockSize.y);


    
    while (0 < iter--) {


            mmult_kernel<<<gridSize, blockSize, 0, streams[i]>>>(d_mat3[i], d_mat1[i], d_mat2[i], nrows, ncols, start_row, end_row);
        }

        for (int i = 0; i < num_gpus; i++) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }

        #ifdef DEBUG
        for (int i = 0; i < num_gpus; i++) {
            hipSetDevice(i);
            long start_row = i * rows_per_gpu + (i < remainder ? i : remainder);
            long end_row = start_row + rows_per_gpu + (i < remainder ? 1 : 0);
            long gpu_nrows = end_row - start_row;
            hipMemcpyAsync(mat3 + start_row * nrows, d_mat3[i], sizeof(float) * gpu_nrows * nrows, hipMemcpyDeviceToHost, streams[i]);
        }
        for (int i = 0; i < num_gpus; i++) {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }
        printmat(mat3, nrows, nrows);
        #endif
    }
    end = clock();

    elapsed_time = ((double)(end - start)) / CLOCKS_PER_SEC;

    // Print the elapsed time
    printf("%f;%d\n", elapsed_time / niter, nrows);



    return 0;
}
